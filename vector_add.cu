//C[i]= A[i]+B[i]
#include <stdio.h>
#include <hip/hip_runtime.h>


#define N 1024

__global__ void vector_add(float *A, float *B, float *C){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    if (i<N){
        C[i]=A[i]+B[i];
    }

}

int main(){
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    float ker, htod, dtoh, total;
    hipEvent_t start, stop;
    hipEvent_t kstart, kstop;
    hipEvent_t hstart, hstop;
    hipEvent_t dstart, dstop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&kstart);hipEventCreate(&hstart);hipEventCreate(&dstart);
    hipEventCreate(&kstop);hipEventCreate(&hstop);hipEventCreate(&dstop);

    hipEventRecord(start);
    size_t size=N*sizeof(float);

    h_A=(float*) malloc(size);
    h_B=(float*) malloc(size);
    h_C=(float*) malloc(size);

    for (int i=0; i<N; i++){
        h_A[i]=i;
        h_B[i]=2*i;
    }
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipEventRecord(hstart);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipEventRecord(hstop);
    hipEventSynchronize(hstop);
    hipEventElapsedTime(&htod, hstart,hstop);
    printf("DTOH Time: %f ms\n",htod);

    int threadsPerBlock =32;
    int blocks = (N+threadsPerBlock-1)/threadsPerBlock;
    hipEventRecord(kstart);
    vector_add<<<blocks,threadsPerBlock>>>(d_A,d_B,d_C);
    hipEventRecord(kstop);
    hipEventSynchronize(kstop);
    hipEventElapsedTime(&ker, kstart,kstop);
    printf("Kernel Execution Time: %f ms\n",ker);

    hipEventRecord(dstart);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipEventRecord(dstop);
    hipEventSynchronize(dstop);
    hipEventElapsedTime(&dtoh, dstart,dstop);
    printf("HTOD Time: %f ms\n",dtoh);
    printf("Total Time: %f\n", ker+dtoh+htod);
    for(int i =0; i<N;i++){
        if (h_C[i]!=h_A[i]+h_B[i]){
            printf("C[%d]=%f\n", i, h_C[i]);
            break;
        }
        if(i+1==N){
            printf("All %d matched\n", N);
        }
    }
    free(h_A);free(h_B);free(h_C);
    hipFree(d_A);hipFree(d_B);hipFree(d_C);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&total, start,stop);
    printf("Total Time: %f ms\n",total);
}